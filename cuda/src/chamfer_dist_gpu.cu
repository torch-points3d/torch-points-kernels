#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <torch/extension.h>
#include <THC/THCAtomics.cuh>
#include "cuda_utils.h"
#include <vector>

template <typename scalar_t>
__global__ void chamfer_dist_kernel(int batch_size, int n, const scalar_t* __restrict__ xyz1, int m,
                                    const scalar_t* __restrict__ xyz2, scalar_t* __restrict__ dist,
                                    int* indexes)
{
    const int batch = 512;
    __shared__ scalar_t buf[batch * 3];
    for (int i = blockIdx.x; i < batch_size; i += gridDim.x)
    {
        for (int k2 = 0; k2 < m; k2 += batch)
        {
            int end_k = min(m, k2 + batch) - k2;
            for (int j = threadIdx.x; j < end_k * 3; j += blockDim.x)
            {
                buf[j] = xyz2[(i * m + k2) * 3 + j];
            }
            __syncthreads();
            for (int j = threadIdx.x + blockIdx.y * blockDim.x; j < n; j += blockDim.x * gridDim.y)
            {
                scalar_t x1 = xyz1[(i * n + j) * 3 + 0];
                scalar_t y1 = xyz1[(i * n + j) * 3 + 1];
                scalar_t z1 = xyz1[(i * n + j) * 3 + 2];
                scalar_t best_dist = 0;
                int best_dist_index = 0;
                int end_ka = end_k - (end_k & 3);
                if (end_ka == batch)
                {
                    for (int k = 0; k < batch; k += 4)
                    {
                        {
                            scalar_t x2 = buf[k * 3 + 0] - x1;
                            scalar_t y2 = buf[k * 3 + 1] - y1;
                            scalar_t z2 = buf[k * 3 + 2] - z1;
                            scalar_t dist = x2 * x2 + y2 * y2 + z2 * z2;

                            if (k == 0 || dist < best_dist)
                            {
                                best_dist = dist;
                                best_dist_index = k + k2;
                            }
                        }
                        {
                            scalar_t x2 = buf[k * 3 + 3] - x1;
                            scalar_t y2 = buf[k * 3 + 4] - y1;
                            scalar_t z2 = buf[k * 3 + 5] - z1;
                            scalar_t dist = x2 * x2 + y2 * y2 + z2 * z2;
                            if (dist < best_dist)
                            {
                                best_dist = dist;
                                best_dist_index = k + k2 + 1;
                            }
                        }
                        {
                            scalar_t x2 = buf[k * 3 + 6] - x1;
                            scalar_t y2 = buf[k * 3 + 7] - y1;
                            scalar_t z2 = buf[k * 3 + 8] - z1;
                            scalar_t dist = x2 * x2 + y2 * y2 + z2 * z2;
                            if (dist < best_dist)
                            {
                                best_dist = dist;
                                best_dist_index = k + k2 + 2;
                            }
                        }
                        {
                            scalar_t x2 = buf[k * 3 + 9] - x1;
                            scalar_t y2 = buf[k * 3 + 10] - y1;
                            scalar_t z2 = buf[k * 3 + 11] - z1;
                            scalar_t dist = x2 * x2 + y2 * y2 + z2 * z2;
                            if (dist < best_dist)
                            {
                                best_dist = dist;
                                best_dist_index = k + k2 + 3;
                            }
                        }
                    }
                }
                else
                {
                    for (int k = 0; k < end_ka; k += 4)
                    {
                        {
                            scalar_t x2 = buf[k * 3 + 0] - x1;
                            scalar_t y2 = buf[k * 3 + 1] - y1;
                            scalar_t z2 = buf[k * 3 + 2] - z1;
                            scalar_t dist = x2 * x2 + y2 * y2 + z2 * z2;
                            if (k == 0 || dist < best_dist)
                            {
                                best_dist = dist;
                                best_dist_index = k + k2;
                            }
                        }
                        {
                            scalar_t x2 = buf[k * 3 + 3] - x1;
                            scalar_t y2 = buf[k * 3 + 4] - y1;
                            scalar_t z2 = buf[k * 3 + 5] - z1;
                            scalar_t dist = x2 * x2 + y2 * y2 + z2 * z2;
                            if (dist < best_dist)
                            {
                                best_dist = dist;
                                best_dist_index = k + k2 + 1;
                            }
                        }
                        {
                            scalar_t x2 = buf[k * 3 + 6] - x1;
                            scalar_t y2 = buf[k * 3 + 7] - y1;
                            scalar_t z2 = buf[k * 3 + 8] - z1;
                            scalar_t dist = x2 * x2 + y2 * y2 + z2 * z2;
                            if (dist < best_dist)
                            {
                                best_dist = dist;
                                best_dist_index = k + k2 + 2;
                            }
                        }
                        {
                            scalar_t x2 = buf[k * 3 + 9] - x1;
                            scalar_t y2 = buf[k * 3 + 10] - y1;
                            scalar_t z2 = buf[k * 3 + 11] - z1;
                            scalar_t dist = x2 * x2 + y2 * y2 + z2 * z2;
                            if (dist < best_dist)
                            {
                                best_dist = dist;
                                best_dist_index = k + k2 + 3;
                            }
                        }
                    }
                }
                for (int k = end_ka; k < end_k; k++)
                {
                    scalar_t x2 = buf[k * 3 + 0] - x1;
                    scalar_t y2 = buf[k * 3 + 1] - y1;
                    scalar_t z2 = buf[k * 3 + 2] - z1;
                    scalar_t dist = x2 * x2 + y2 * y2 + z2 * z2;
                    if (k == 0 || dist < best_dist)
                    {
                        best_dist = dist;
                        best_dist_index = k + k2;
                    }
                }
                if (k2 == 0 || dist[(i * n + j)] > best_dist)
                {
                    dist[(i * n + j)] = best_dist;
                    indexes[(i * n + j)] = best_dist_index;
                }
            }
            __syncthreads();
        }
    }
}

std::vector<torch::Tensor> chamfer_dist_kernel_wrapper(torch::Tensor xyz1, torch::Tensor xyz2)
{
    const int batch_size = xyz1.size(0);
    const int n = xyz1.size(1); // num_points point cloud A
    const int m = xyz2.size(1); // num_points point cloud B
    auto dist1 = torch::zeros({batch_size, n}, torch::CUDA(xyz1.scalar_type()));
    auto dist2 = torch::zeros({batch_size, m}, torch::CUDA(xyz1.scalar_type()));
    auto idx1 = torch::zeros({batch_size, n}, torch::CUDA(torch::kInt));
    auto idx2 = torch::zeros({batch_size, m}, torch::CUDA(torch::kInt));

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
        xyz1.scalar_type(), "chamfer_dist_cuda", ([&] {
            chamfer_dist_kernel<scalar_t><<<dim3(32, 16, 1), 512>>>(
                batch_size, n, xyz1.data_ptr<scalar_t>(), m, xyz2.data_ptr<scalar_t>(),
                dist1.data_ptr<scalar_t>(), idx1.data_ptr<int>());

            chamfer_dist_kernel<scalar_t><<<dim3(32, 16, 1), 512>>>(
                batch_size, m, xyz2.data_ptr<scalar_t>(), n, xyz1.data_ptr<scalar_t>(),
                dist2.data_ptr<scalar_t>(), idx2.data_ptr<int>());
        }));

    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("Error in chamfer_dist_kernel_wrapper: %s\n", hipGetErrorString(err));
    }
    return {dist1, dist2, idx1, idx2};
}

template <typename scalar_t>
__global__ void chamfer_dist_grad_kernel(int b, int n, const scalar_t* __restrict__ xyz1, int m,
                                         const scalar_t* __restrict__ xyz2,
                                         const scalar_t* __restrict__ grad_dist1, const int* idx1,
                                         scalar_t* __restrict__ grad_xyz1,
                                         scalar_t* __restrict__ grad_xyz2)
{
    for (int i = blockIdx.x; i < b; i += gridDim.x)
    {
        for (int j = threadIdx.x + blockIdx.y * blockDim.x; j < n; j += blockDim.x * gridDim.y)
        {
            scalar_t x1 = xyz1[(i * n + j) * 3 + 0];
            scalar_t y1 = xyz1[(i * n + j) * 3 + 1];
            scalar_t z1 = xyz1[(i * n + j) * 3 + 2];
            int j2 = idx1[i * n + j];
            scalar_t x2 = xyz2[(i * m + j2) * 3 + 0];
            scalar_t y2 = xyz2[(i * m + j2) * 3 + 1];
            scalar_t z2 = xyz2[(i * m + j2) * 3 + 2];
            scalar_t g = grad_dist1[i * n + j] * 2;
            gpuAtomicAdd(&(grad_xyz1[(i * n + j) * 3 + 0]), g * (x1 - x2));
            gpuAtomicAdd(&(grad_xyz1[(i * n + j) * 3 + 1]), g * (y1 - y2));
            gpuAtomicAdd(&(grad_xyz1[(i * n + j) * 3 + 2]), g * (z1 - z2));
            gpuAtomicAdd(&(grad_xyz2[(i * m + j2) * 3 + 0]), -(g * (x1 - x2)));
            gpuAtomicAdd(&(grad_xyz2[(i * m + j2) * 3 + 1]), -(g * (y1 - y2)));
            gpuAtomicAdd(&(grad_xyz2[(i * m + j2) * 3 + 2]), -(g * (z1 - z2)));
        }
    }
}

std::vector<torch::Tensor> chamfer_dist_grad_kernel_wrapper(torch::Tensor xyz1, torch::Tensor xyz2,
                                                            torch::Tensor idx1, torch::Tensor idx2,
                                                            torch::Tensor grad_dist1,
                                                            torch::Tensor grad_dist2)
{
    const int batch_size = xyz1.size(0);
    const int n = xyz1.size(1); // num_points point cloud A
    const int m = xyz2.size(1); // num_points point cloud B
    auto grad_xyz1 = torch::zeros_like(xyz1);
    auto grad_xyz2 = torch::zeros_like(xyz2);

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
        xyz1.scalar_type(), "chamfer_dist_grad_cuda", ([&] {
            chamfer_dist_grad_kernel<scalar_t><<<dim3(1, 16, 1), 256>>>(
                batch_size, n, xyz1.data_ptr<scalar_t>(), m, xyz2.data_ptr<scalar_t>(),
                grad_dist1.data_ptr<scalar_t>(), idx1.data_ptr<int>(),
                grad_xyz1.data_ptr<scalar_t>(), grad_xyz2.data_ptr<scalar_t>());

            chamfer_dist_grad_kernel<scalar_t><<<dim3(1, 16, 1), 256>>>(
                batch_size, m, xyz2.data_ptr<scalar_t>(), n, xyz1.data_ptr<scalar_t>(),
                grad_dist2.data_ptr<scalar_t>(), idx2.data_ptr<int>(),
                grad_xyz2.data_ptr<scalar_t>(), grad_xyz1.data_ptr<scalar_t>());
        }));

    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("Error in chamfer_dist_grad_kernel_wrapper: %s\n", hipGetErrorString(err));
    }
    return {grad_xyz1, grad_xyz2};
}
