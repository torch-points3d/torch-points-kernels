#include "hip/hip_runtime.h"
#include "cuda_utils.h"
#include <THC/THCAtomics.cuh>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <torch/extension.h>
#include <vector>

// input: unknown(b, n, 3) known(b, m, 3)
// output: dist2(b, n, 3), idx(b, n, 3)
template <typename scalar_t>
__global__ void three_nn_kernel(int b, int n, int m, const double upper_bd,
                                const scalar_t* __restrict__ unknown,
                                const scalar_t* __restrict__ known, scalar_t* __restrict__ dist2,
                                int* __restrict__ idx)
{
    int batch_index = blockIdx.x;
    unknown += batch_index * n * 3;
    known += batch_index * m * 3;
    dist2 += batch_index * n * 3;
    idx += batch_index * n * 3;

    int index = threadIdx.x;
    int stride = blockDim.x;
    for (int j = index; j < n; j += stride)
    {
        scalar_t ux = unknown[j * 3 + 0];
        scalar_t uy = unknown[j * 3 + 1];
        scalar_t uz = unknown[j * 3 + 2];
        scalar_t best1 = upper_bd, best2 = upper_bd, best3 = upper_bd;

        int besti1 = 0, besti2 = 0, besti3 = 0;
        for (int k = 0; k < m; ++k)
        {
            scalar_t x = known[k * 3 + 0];
            scalar_t y = known[k * 3 + 1];
            scalar_t z = known[k * 3 + 2];
            scalar_t d = (ux - x) * (ux - x) + (uy - y) * (uy - y) + (uz - z) * (uz - z);
            if (d < best1)
            {
                best3 = best2;
                besti3 = besti2;
                best2 = best1;
                besti2 = besti1;
                best1 = d;
                besti1 = k;
            }
            else if (d < best2)
            {
                best3 = best2;
                besti3 = besti2;
                best2 = d;
                besti2 = k;
            }
            else if (d < best3)
            {
                best3 = d;
                besti3 = k;
            }
        }
        dist2[j * 3 + 0] = best1;
        dist2[j * 3 + 1] = best2;
        dist2[j * 3 + 2] = best3;

        idx[j * 3 + 0] = besti1;
        idx[j * 3 + 1] = besti2;
        idx[j * 3 + 2] = besti3;
    }
}

std::vector<torch::Tensor> three_nn_kernel_wrapper(torch::Tensor unknowns, torch::Tensor knows)
{
    int b = unknowns.size(0);
    int n = unknowns.size(1);
    int m = knows.size(1);

    auto idx = torch::zeros({b, n, 3}, torch::CUDA(torch::kInt));
    auto dist2 = torch::zeros({b, n, 3}, torch::CUDA(unknowns.scalar_type()));
    double upper_bd = 0;
    switch (unknowns.scalar_type())
    {
    case torch::ScalarType::Double:
        upper_bd = 1e40;
        break;
    case torch::ScalarType::Half:
        upper_bd = 65504;
        break;
    default:
        upper_bd = 1e20;
        break;
    }
    hipStream_t stream = at::cuda::getCurrentCUDAStream();
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
        unknowns.scalar_type(), "three_nn_kernel_cuda",
        (
            [&]
            {
                three_nn_kernel<scalar_t><<<b, opt_n_threads(n), 0, stream>>>(
                    b, n, m, upper_bd, unknowns.data_ptr<scalar_t>(), knows.data_ptr<scalar_t>(),
                    dist2.data_ptr<scalar_t>(), idx.data_ptr<int>());
            }));
    CUDA_CHECK_ERRORS();
    return {dist2, idx};
}

// input: points(b, c, m), idx(b, n, 3), weight(b, n, 3)
// output: out(b, c, n)
template <typename scalar_t>
__global__ void
three_interpolate_kernel(int b, int c, int m, int n, const scalar_t* __restrict__ points,
                         const int* __restrict__ idx, const scalar_t* __restrict__ weight,
                         scalar_t* __restrict__ out)
{
    int batch_index = blockIdx.x;
    points += batch_index * m * c;

    idx += batch_index * n * 3;
    weight += batch_index * n * 3;

    out += batch_index * n * c;

    const int index = threadIdx.y * blockDim.x + threadIdx.x;
    const int stride = blockDim.y * blockDim.x;
    for (int i = index; i < c * n; i += stride)
    {
        const int l = i / n;
        const int j = i % n;
        scalar_t w1 = weight[j * 3 + 0];
        scalar_t w2 = weight[j * 3 + 1];
        scalar_t w3 = weight[j * 3 + 2];

        int i1 = idx[j * 3 + 0];
        int i2 = idx[j * 3 + 1];
        int i3 = idx[j * 3 + 2];

        out[i] = points[l * m + i1] * w1 + points[l * m + i2] * w2 + points[l * m + i3] * w3;
    }
}

torch::Tensor three_interpolate_kernel_wrapper(torch::Tensor points, torch::Tensor idx,
                                               torch::Tensor weight)
{
    int b = points.size(0);
    int c = points.size(1);
    int m = points.size(2);
    int n = idx.size(1);

    auto out = torch::zeros({b, c, n}, torch::CUDA(points.scalar_type()));

    hipStream_t stream = at::cuda::getCurrentCUDAStream();
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
        points.scalar_type(), "three_interpolate_kernel_cuda",
        (
            [&]
            {
                three_interpolate_kernel<scalar_t><<<b, opt_block_config(n, c), 0, stream>>>(
                    b, c, m, n, points.data_ptr<scalar_t>(), idx.data_ptr<int>(),
                    weight.data_ptr<scalar_t>(), out.data_ptr<scalar_t>());
            }));

    CUDA_CHECK_ERRORS();
    return out;
}

// input: grad_out(b, c, n), idx(b, n, 3), weight(b, n, 3)
// output: grad_points(b, c, m)
template <typename scalar_t>
__global__ void
three_interpolate_grad_kernel(int b, int c, int n, int m, const scalar_t* __restrict__ grad_out,
                              const int* __restrict__ idx, const scalar_t* __restrict__ weight,
                              scalar_t* __restrict__ grad_points)
{
    int batch_index = blockIdx.x;
    grad_out += batch_index * n * c;
    idx += batch_index * n * 3;
    weight += batch_index * n * 3;
    grad_points += batch_index * m * c;

    const int index = threadIdx.y * blockDim.x + threadIdx.x;
    const int stride = blockDim.y * blockDim.x;
    for (int i = index; i < c * n; i += stride)
    {
        const int l = i / n;
        const int j = i % n;
        scalar_t w1 = weight[j * 3 + 0];
        scalar_t w2 = weight[j * 3 + 1];
        scalar_t w3 = weight[j * 3 + 2];

        int i1 = idx[j * 3 + 0];
        int i2 = idx[j * 3 + 1];
        int i3 = idx[j * 3 + 2];

        gpuAtomicAdd(grad_points + l * m + i1, grad_out[i] * w1);
        gpuAtomicAdd(grad_points + l * m + i2, grad_out[i] * w2);
        gpuAtomicAdd(grad_points + l * m + i3, grad_out[i] * w3);
    }
}

torch::Tensor three_interpolate_grad_kernel_wrapper(torch::Tensor grad_out, torch::Tensor idx,
                                                    torch::Tensor weight, const int m)
{
    int b = grad_out.size(0);
    int c = grad_out.size(1);
    int n = grad_out.size(2);

    auto grad_points = torch::zeros({b, c, m}, torch::CUDA(grad_out.scalar_type()));

    hipStream_t stream = at::cuda::getCurrentCUDAStream();
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
        grad_out.scalar_type(), "three_interpolate_grad_kernel_cuda",
        (
            [&]
            {
                three_interpolate_grad_kernel<scalar_t><<<b, opt_block_config(n, c), 0, stream>>>(
                    b, c, n, m, grad_out.data_ptr<scalar_t>(), idx.data_ptr<int>(),
                    weight.data_ptr<scalar_t>(), grad_points.data_ptr<scalar_t>());
            }));
    CUDA_CHECK_ERRORS();
    return grad_points;
}
